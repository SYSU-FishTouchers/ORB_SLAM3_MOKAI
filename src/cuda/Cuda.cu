#include <hip/hip_runtime_api.h>
#include <cuda/Cuda.hpp>

namespace ORB_SLAM3 { namespace cuda {
  void deviceSynchronize() {
    checkCudaErrors( hipDeviceSynchronize() );
  }
} }
